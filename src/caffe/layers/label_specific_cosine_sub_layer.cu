#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/label_specific_cosine_sub_layer.hpp"

namespace caffe {


  template <typename Dtype>
  __global__ void LabelSpecificCosineSubForward(const int n, const int dim, const Dtype* label,
                                                const Dtype* bottom_data, Dtype* top_data,
                                                const Dtype cos_margin_theta, const Dtype sin_margin_theta,
                                                const Dtype sq_margin_theta , const Dtype margin_m ,
                                                const Dtype threshold) {
    CUDA_KERNEL_LOOP(i, n) {
      const int gt = static_cast<int>(label[i]);
      const int idx = i * dim + gt;
      const Dtype cos_theta = bottom_data[idx];
      Dtype tmp = cos_theta;
      if (cos_theta > threshold)
      {
           const Dtype sq_cos_theta = cos_theta*cos_theta;
           if ((sq_cos_theta <= (Dtype)1.) && (sq_cos_theta >= (Dtype)-1.)){
               const Dtype sin_theta = sqrt((Dtype)1. - sq_cos_theta);
               tmp = cos_theta*cos_margin_theta - sin_theta*sin_margin_theta;  //modified by zkx 2018-03-23
           }
      } else {
          tmp = cos_theta - sq_margin_theta;
      }

      if (tmp > (-margin_m))
          tmp += margin_m;

      top_data[idx] = tmp;
    }
  }

  template <typename Dtype>
  void LabelSpecificCosineSubLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    const int num = bottom[0]->num();
    const int count = bottom[0]->count();
    const int dim = count / num;

    if (top[0] != bottom[0]) caffe_copy(count, bottom_data, top_data);
    if (this->phase_ == TEST) return;

    if (top.size() == 2) {
      top[1]->mutable_cpu_data()[0] = margin_theta;
    }

    // NOLINT_NEXT_LINE(whitespace/operators)
    LabelSpecificCosineSubForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, label_data, bottom_data, top_data, cos_margin_theta, sin_margin_theta,sq_margin_theta,margin_m,threshold);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  __global__ void LabelSpecificCosineSubBackward(const int n, const int dim, const Dtype* label,
                                                const Dtype* bottom_data, const Dtype* top_diff,
                                                Dtype* bottom_diff, const Dtype cos_margin_theta,
                                                const Dtype sin_margin_theta, const Dtype threshold) {
    CUDA_KERNEL_LOOP(i, n) {
      const int gt = static_cast<int>(label[i]);
      const int idx = i * dim + gt;
      const Dtype cos_theta = bottom_data[idx];
      //clip cos_theta =[-1,1]
      if (cos_theta > threshold)
      {
          const Dtype sq_cos_theta = cos_theta*cos_theta;
          if ((sq_cos_theta <= (Dtype)1.) && (sq_cos_theta >= (Dtype)-1.)){
              const Dtype sin_theta = sqrt((Dtype)1. - sq_cos_theta) + (Dtype)1e-6;
              bottom_diff[idx] = (cos_margin_theta + sin_margin_theta*(cos_theta/sin_theta))*top_diff[idx]; //modified by zkx 2018-03-23
          }
      }

    }
  }

  template <typename Dtype>
  void LabelSpecificCosineSubLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                     const vector<bool>& propagate_down,
                                                     const vector<Blob<Dtype>*>& bottom) {
    if (top[0] != bottom[0] && propagate_down[0]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const int count = bottom[0]->count();
      caffe_copy(count, top_diff, bottom_diff);

      const Dtype* bottom_data = bottom[0]->gpu_data();
      const Dtype* label_data = bottom[1]->gpu_data();
      const int num = bottom[0]->num();
      const int dim = count / num;
      // NOLINT_NEXT_LINE(whitespace/operators)
      LabelSpecificCosineSubBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, label_data, bottom_data, top_diff, bottom_diff, cos_margin_theta, sin_margin_theta, threshold);
      CUDA_POST_KERNEL_CHECK;
    }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(LabelSpecificCosineSubLayer);
}  // namespace caffe
